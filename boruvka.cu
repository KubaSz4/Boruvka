
#include <hip/hip_runtime.h>
#include <cstdio>
#include <algorithm>
#include <climits>

extern "C" {

	int min(int a, int b){
		return a < b ? a : b; 
	}

	__global__
    void set(int* parent, int* newParent, int *res, int n){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;

        if(thid >= n) return;

        parent[thid] = newParent[thid] = thid;
        res[thid] = 0;
    }


    //one thread - one vertex
    //find the closest neighbour for each vertex
	__global__
	void closest_neighbour(int* G, int* parent, int* newParent, int* roundNeighbourDist, int* toSubtract, int n){
		int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
		
        if (thid >= n) return;
        if (parent[thid] != thid) return;

        int minimum = INT_MAX;
        int neighbour = -1;

		for(int i=0; i<n; i++){
            int dist = G[thid+i*n];
            if(dist >= minimum) continue;

            minimum = dist;
            neighbour = parent[i];
        }

        if (neighbour == -1) return;

        toSubtract[thid] = roundNeighbourDist[thid] = minimum;


        //start vertex union
        while(true){
            while(thid != newParent[thid]) thid = newParent[thid];
            while(neighbour != newParent[neighbour]) neighbour = newParent[neighbour];
            if(thid < neighbour){
                if (neighbour == atomicCAS(&newParent[neighbour], neighbour, thid)) {
                    break;
                }
            }
            else{
                if (thid == atomicCAS(&newParent[thid], thid, neighbour)){
                    break;
                } 
            }
        }
    }

    //one thread - one vertex
    //find component root for every vertex
    //sum MST for component in root
    __global__
    void unionn(int* parent, int* newParent, int* MSTres, int* roundNeighbourDist, int* toSubtract, int n, int* goOn){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        
        if (thid >= n) return;

        int parentToBe = newParent[parent[thid]];

        while(parentToBe != newParent[parentToBe]) parentToBe = newParent[parentToBe];

        if(thid == parent[thid] && parentToBe != thid){ //previously root, now part of new component
            atomicAdd(&MSTres[parentToBe], MSTres[thid]);
            atomicAdd(&MSTres[parentToBe], roundNeighbourDist[thid]);
            atomicMin(&toSubtract[parentToBe], roundNeighbourDist[thid]);
        }
        else if(parentToBe == thid){ //new component root
            atomicAdd(&MSTres[parentToBe], roundNeighbourDist[thid]);
            atomicMin(&toSubtract[parentToBe], roundNeighbourDist[thid]);
        }

        newParent[thid] = parent[thid] = parentToBe;

        //finally all vertices should belong to component 0        
        if(parent[thid] != 0){
            *goOn = 1;
        }
    }

    //subtract the repeated root in each new component
    __global__
    void subtract(int *parent, int * MSTres, int* toSubtract, int n){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        
        if (thid >= n) return;
        if (parent[thid] != thid) return;

        MSTres[thid] -= toSubtract[thid];
    }

    __global__
    void transpose(int* A, int* AT, int N){
        int M = N;
        int thidx = (blockIdx.x * blockDim.x) + threadIdx.x;
        int thidy = (blockIdx.y * blockDim.y) + threadIdx.y;
        
        __shared__ int T[32][33];
        
        if(thidx < M & thidy < N)
            T[threadIdx.y][threadIdx.x] = A[thidy*M+thidx];
        __syncthreads();


        thidx = (blockIdx.y * blockDim.x) + threadIdx.x;
        thidy = (blockIdx.x * blockDim.y) + threadIdx.y;
        
        if(thidx < N & thidy < M)
            AT[thidy*N+thidx] = T[threadIdx.x][threadIdx.y];
    }

    //one thread - one vertex (destination)
    //choose the smallest distance from each component to the vertex
    __global__
    void merge(int *G, int *parent, int n){
        int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
        
        if (thid >= n) return;

        int thidGroup = parent[thid];

        for(int i=0; i<n; i++){
            if(i == thid){ //set distance to self as INT_MAX
                G[thidGroup*n+thid] = INT_MAX;
            }
            else{
                int iGroup = parent[i];
                if(iGroup != thidGroup){
                    G[iGroup*n+thid] = min(G[iGroup*n+thid], G[i*n+thid]);
                }                
            }
        }
    }
}